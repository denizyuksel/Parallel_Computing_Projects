#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <limits.h>


#define M_PI 3.1415926535897
#define VECTOR_COUNT 2

hipError_t computeElementsHelper(int* a, int* b, int* lengthNoSqrt, int* dotProduct, int N, int blockSize);

__global__ void computeElementsKernel(int* lengthNoSqrt, int* product, int* a, int* b, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        //printf("Doing something with thread %d\n", i);
        //printf("Element: %d %d\n", a[i], b[i]);

        //find the dot product.
        atomicAdd(product, a[i] * b[i]);

        //printf("Sumsquares one before: %d\n", lengthNoSqrt[0]);
        //printf("Sumsquares two before: %d\n", lengthNoSqrt[1]);

        atomicAdd(&(lengthNoSqrt[0]), a[i] * a[i]);
        atomicAdd(&(lengthNoSqrt[1]), b[i] * b[i]);


        //printf("Sumsquares one after: %d\n", lengthNoSqrt[0]);
        //printf("Sumsquares two after: %d\n", lengthNoSqrt[1]);

    }

}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int* genVector(int N) {
    int* vector = (int*)malloc(sizeof(int) * N);

    for (int i = 0; i < N; i++) {
        int randNum = rand() % 20 - 10;
        vector[i] = randNum;
    }
    return vector;
} 

int findDotProduct(int* a, int* b, int N) {
    int sum = 0;
    for (int i = 0; i < N; i++) {
        sum = sum + (a[i] * b[i]);
    }
    return sum;
}

void printArray(int* x, int size) {
    for (int i = 0; i < size; i++) {
        printf("arr[%d] = %d\n", i, x[i]);
    }
}

double findVectorLength(int* x, int N) {
    int sumSquares = 0;
    for (int i = 0; i < N; i++) {
        sumSquares = sumSquares + pow(x[i], 2);
    }
    //printf("SumSquares serial: %d\n", sumSquares);
    double distance = sqrt(sumSquares);
    return distance;
    
}

double convertToDegrees(double rad) {
   return rad * (180 / M_PI);
}

void printDeviceProperties() {
    printf("--------------------DEVICE PROPERTIES----------------------\n\n");

    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }

}

double doTheSerialThing(int* vectorOne, int* vectorTwo, int SIZE) {
    //printf("-----------------SERIAL IMPLEMENTATION----------------------\n\n");
    double dotProduct = (double)findDotProduct(vectorOne, vectorTwo, SIZE);
    double vectorLengthOne = findVectorLength(vectorOne, SIZE);
    double vectorLengthTwo = findVectorLength(vectorTwo, SIZE);

    double cosTheta = dotProduct / (vectorLengthOne * vectorLengthTwo);
    double angleInRadians = acos(cosTheta);
    double angleInDegrees = convertToDegrees(angleInRadians);
    //printf("length one: %f\n", vectorLengthOne);
    //printf("length two: %f\n", vectorLengthTwo);

    //printf("Angle in radians: %f\n", angleInRadians);

    //printArray(vectorOne, SIZE);
    //printArray(vectorTwo, SIZE);

    //printf("DOT PRODUCT SERIAL: %f\n", dotProduct);
    return angleInDegrees;
}

int main(int argc, char** argv)
{
    //Before beginning, print device properties.
    //printDeviceProperties();

    srand(time(NULL));
    
    clock_t start, end;
    double cpu_time_used;

    int SIZE = atoi(argv[1]);
    int BLOCK_SIZE = atoi(argv[2]);
    int* vectorOne = NULL;
    int* vectorTwo = NULL;
    int lengthsNoSqrt[VECTOR_COUNT] = { 0 };
    int dotProduct[1] = { 0 };
    double angleSerial = 0;

    int numberBlocks = 0;

    if (SIZE % BLOCK_SIZE == 0)
        numberBlocks = SIZE / BLOCK_SIZE;
    else
        numberBlocks = (SIZE / BLOCK_SIZE) + 1;

    printf("Info\n------------------\n");
    printf("Number of elements: %d\n", SIZE);
    printf("Number of threads per block: %d\n", BLOCK_SIZE);
    printf("Number of blocks will be created: %d\n\n", numberBlocks);
   


    //arrays will be generated
    if (argc == 3) {
        printf("Time\n------------------\n");

        start = clock();              
        vectorOne = genVector(SIZE);
        vectorTwo = genVector(SIZE);
        end = clock();
        cpu_time_used = ((double)(end - start) * 1000) / CLOCKS_PER_SEC;

        printf("Time for the array generation : %f ms\n", cpu_time_used);
    }

    //arrays will be read from file.
    else if (argc == 4) {

        char const* const fileName = argv[3]; /* should check that argc > 1 */
        FILE* file = fopen(fileName, "r"); /* should check the result */
        char line[256];

        fgets(line, sizeof(line), file);
        int count = atoi(line);
        int* allArray = (int*)malloc(sizeof(int) * count * 2);
        vectorOne = (int*)malloc(sizeof(int) * count);
        vectorTwo = (int*)malloc(sizeof(int) * count);

        int i = 0;
        //printf("COUNT: %d\n", count);
        while (fgets(line, sizeof(line), file)) {
            /* note that fgets don't strip the terminating \n, checking its
               presence would allow to handle lines longer that sizeof(line) */
            int number = atoi(line);
            allArray[i] = number;
            i++;
        }
        /* may check feof here to make a difference between eof and io failure -- network
           timeout for instance */

        /*
        for (int i = 0; i < count; i++) {
            printf("allArray[%d] = %d\n", i, allArray[i]);
        }
        */
        
        for (int i = 0; i < count; i++) {
            vectorOne[i] = allArray[i];
        }
        for (int i = count; i < count * 2; i++) {
            vectorTwo[i - count] = allArray[i];
        }     
        fclose(file);
    }

    else {
        printf("GIVE APPROPRIATE NUMBER OF ARGUMENTS PLEASE!!!\n");
        return 0;
    }



    start = clock();
    angleSerial = doTheSerialThing(vectorOne, vectorTwo, SIZE);
    end = clock();
    cpu_time_used = ((double)(end - start) * 1000) / CLOCKS_PER_SEC;

    printf("Time for the CPU function: %f ms\n", cpu_time_used);

    //printf("---------------------PARALLEL IMPLEMENTATION-----------------\n\n");

    // Calculate angle with CUDA.
    hipError_t cudaStatus = computeElementsHelper(vectorOne, vectorTwo, lengthsNoSqrt, dotProduct, SIZE, BLOCK_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "computeElements failed!");
        return 1;
    }

    // find the angle here.
    double lenOne = sqrt( (double) lengthsNoSqrt[0]);
    double lenTwo = sqrt( (double) lengthsNoSqrt[1]);
    double cosTheta = (  ((double) (dotProduct[0])) / (lenOne * lenTwo));
    double angleInRadians = acos(cosTheta);
    double angle = convertToDegrees(angleInRadians);

    printf("\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    printf("Results\n-----------------\n");
    printf("CPU Result: %0.3f\n", angleSerial);
    printf("GPU Result: %0.3f\n", angle);

 
    return 0;
}


hipError_t computeElementsHelper(int* a, int* b, int* lengthNoSqrt, int* dotProduct, int N, int blockSize)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_lengthNoSqrt = 0;
    int* dev_product = 0;
    hipError_t cudaStatus;

    clock_t start, end;
    double timeUsed;
    double totalGpuTime = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
    }

    cudaStatus = hipMalloc((void**)&dev_a, N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev a failed!\n");
    }

    cudaStatus = hipMalloc((void**)&dev_b, N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev b failed!\n");
    }

    cudaStatus = hipMalloc((void**)&dev_lengthNoSqrt, VECTOR_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev length failed!\n");
    }

    cudaStatus = hipMalloc((void**)&dev_product, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev product failed!\n");
    }

    // Copy input vectors from host memory to GPU buffers.

    start = clock();

    cudaStatus = hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
    }

    cudaStatus = hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
    }

    end = clock();
    timeUsed = ((double)(end - start) * 1000) / CLOCKS_PER_SEC;
    totalGpuTime += timeUsed;

    printf("Time for the Host to Device transfer: %f ms\n", timeUsed);
    

    // Launch a kernel on the GPU with one thread for each element.
    int numberBlocks = 0;

    if (N % blockSize == 0)
        numberBlocks = N / blockSize;
    else
        numberBlocks = (N / blockSize) + 1;

    start = clock();

    computeElementsKernel <<< numberBlocks, blockSize >>> (dev_lengthNoSqrt, dev_product, dev_a, dev_b, N);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "computeElementsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }    

    end = clock();
    timeUsed = ((double)(end - start) * 1000) / CLOCKS_PER_SEC;
    totalGpuTime += timeUsed;

    printf("Time for the kernel execution: %f ms\n", timeUsed);

    start = clock();

    cudaStatus = hipMemcpy(lengthNoSqrt, dev_lengthNoSqrt, VECTOR_COUNT * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy for dev_lengths failed!\n");
    }

    cudaStatus = hipMemcpy(dotProduct, dev_product, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy for dotProduct failed!\n");
    }

    end = clock();
    timeUsed = ((double)(end - start) * 1000) / CLOCKS_PER_SEC;
    printf("Time for the Device to Host transfer: %f ms\n", timeUsed);
    totalGpuTime += timeUsed;

    printf("Total execution time for GPU: %f ms\n", totalGpuTime);

    hipFree(dev_product);
    hipFree(dev_lengthNoSqrt);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}